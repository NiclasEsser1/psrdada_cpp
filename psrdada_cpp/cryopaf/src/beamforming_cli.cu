#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/complex.h>
#include <hip/hip_runtime.h>
#include <random>
#include <cmath>
#include <fstream>
#include <chrono>
#include <unordered_map>


#include "boost/filesystem.hpp"
#include "boost/program_options.hpp"

#include "psrdada_cpp/cli_utils.hpp"
#include "psrdada_cpp/dada_input_stream.hpp"
#include "psrdada_cpp/dada_output_stream.hpp"
#include "psrdada_cpp/multilog.hpp"

#include "psrdada_cpp/cryopaf/Pipeline.cuh"


const size_t ERROR_IN_COMMAND_LINE = 1;
const size_t SUCCESS = 0;
const size_t ERROR_UNHANDLED_EXCEPTION = 2;

using namespace psrdada_cpp;
using namespace psrdada_cpp::cryopaf;

template<typename T>
void launch(PipelineConfig& conf)
{
    MultiLog log(conf.logname);
    DadaOutputStream output (conf.out_key, log);
    if (conf.mode == "voltage")
    {
      // For voltage beamformer pipeline ComputeType and ResultType has to be equal
      Pipeline<decltype(output), T, T> pipeline(conf, log, output);
      DadaInputStream<decltype(pipeline)> input(conf.in_key, log, pipeline);
      input.start();
    }
    else if (conf.mode == "power")
    {
      // For power beamformer pipeline ResultType is ComputeType::x
      Pipeline<decltype(output), T, decltype(T::x)> pipeline(conf, log, output);
      DadaInputStream<decltype(pipeline)> input(conf.in_key, log, pipeline);
      input.start();
    }
    else
    {
      BOOST_LOG_TRIVIAL(error) << "Beamform mode " << conf.mode << " not implemented";
    }
}

int main(int argc, char** argv)
{
    try
    {
        // Variables to store command line options
        PipelineConfig conf;
        std::string precision;
        std::string kind;

        // Parse command line
        namespace po = boost::program_options;
        po::options_description desc("Options");
        desc.add_options()
        ("help,h", "Print help messages")
        ("in_key", po::value<std::string>()->required()
          ->notifier([&conf](std::string key){conf.in_key = string_to_key(key);}), "Input dada key")
        ("out_key", po::value<std::string>()->required()
          ->notifier([&conf](std::string key){conf.out_key = string_to_key(key);}), "Output dada key")
        ("samples", po::value<std::size_t>(&conf.n_samples)->default_value(262144), "Number of samples within one dada block")
        ("channels", po::value<std::size_t>(&conf.n_channel)->default_value(7), "Number of channels")
        ("elements", po::value<std::size_t>(&conf.n_elements)->default_value(36), "Number of elments")
        ("beams", po::value<std::size_t>(&conf.n_beam)->default_value(36), "Number of beams")
        ("integration", po::value<std::size_t>(&conf.integration)->default_value(1), "Integration interval; must be multiple 2^n and smaller 32")
        ("device", po::value<int>(&conf.device_id)->default_value(0), "ID of GPU device")
        ("mode", po::value<std::string>(&conf.mode)->default_value("power"), "Beamforming mode; valid inputs 'power' and 'voltage'")
        ("input_type", po::value<std::string>(&conf.input_type)->default_value("float"), "Data type of received input data (supported types: float). It is always expected that the samples are complex (e.g. internally float becomes float2)")
        ("precision", po::value<std::string>(&precision)->default_value("single"), "Compute type of GEMM operation; supported precisions 'half' and 'single'")
        ("protocol", po::value<std::string>(&conf.protocol)->default_value("codif"), "Protocol of input data; supported 'codif', 'spead' and 'dummy_input'.")
        ("log", po::value<std::string>(&conf.logname)->default_value("cryo_beamform.log"), "Directory of logfile");

        po::variables_map vm;
        try
        {
            po::store(po::parse_command_line(argc, argv, desc), vm);
            if ( vm.count("help")  )
            {
                std::cout << "Cryopaf -- The CryoPAF Controller implementations" << std::endl
                << desc << std::endl;
                return SUCCESS;
            }
            po::notify(vm);
        }
        catch(po::error& e)
        {
            std::cerr << "ERROR: " << e.what() << std::endl << std::endl;
            std::cerr << desc << std::endl;
            return ERROR_IN_COMMAND_LINE;
        }
        if(precision == "half")
        {
          launch<__half2>(conf);
        }
        else if(precision == "single")
        {
          launch<float2>(conf);
        }
        else
        {
          BOOST_LOG_TRIVIAL(error) << "Compute type " << precision << " not implemented";
        }
    }
    catch(std::exception& e)
    {
        std::cerr << "Unhandled Exception reached the top of main: "
            << e.what() << ", application will now exit" << std::endl;
        return ERROR_UNHANDLED_EXCEPTION;
    }
    return SUCCESS;
}
