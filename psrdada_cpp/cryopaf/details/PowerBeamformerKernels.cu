
#include <hip/hip_runtime.h>
#ifdef POWER_BEAMFORMER_CUH_



namespace psrdada_cpp{
namespace cryopaf{
namespace beamforming{



/** Final kernel **/

template<typename T=__half2, typename U=__half>
__global__ void coherent_bf_power(const __half2 *idata, __half *odata, const __half2 *weight)
{
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x; // Beams
    const int bidy = blockIdx.y; // Channels
    const int warp_idx = tidx % WARP_SIZE;
    const int warp = tidx / WARP_SIZE;


    int tscrunch_cnt = 0;

    const int idata_glob_offset = bidy * N_ELEMENTS_CB;
    const int weight_glob_offset = bidx * WARPS_CB * N_CHANNEL_CB * N_ELEMENTS_CB + bidy * N_ELEMENTS_CB;
    int odata_glob_idx = (bidx * WARPS_CB + warp) * N_CHANNEL_CB * N_TIMESTAMPS_OUT_CB + bidy * N_TIMESTAMPS_OUT_CB;

    __shared__ __half2 s_input[N_ELEMENTS_CB];
    __shared__ __half2 s_weight[WARPS_CB][N_ELEMENTS_CB];
    __shared__ __half2 s_inter[WARPS_CB][WARP_SIZE];
    __shared__ __half s_output[WARPS_CB][WARP_SIZE];


    for(int b = 0; b < WARPS_CB; b++)
    {
        int weight_glob_idx = weight_glob_offset + b * N_CHANNEL_CB * N_ELEMENTS_CB;
        for(int a = tidx; a < N_ELEMENTS_CB; a+=N_THREAD_CB)
        {
            s_weight[b][a] = weight[weight_glob_idx + a];
        }
    }

    for(int t = 0; t < N_TIMESTAMPS_CB; t++)
    {
        int idata_glob_idx = idata_glob_offset + t * N_ELEMENTS_CB * N_CHANNEL_CB;
        for(int a = tidx; a < N_ELEMENTS_CB; a+=N_THREAD_CB)
        {
            // All threads loading raw voltages
            s_input[a] = idata[idata_glob_idx + a];
        }
		__half2 voltage = {0,0};
        __syncthreads();
        // Each warp processes complex mulitplications for its beam
        for(int a = warp_idx; a < N_ELEMENTS_CB; a+=WARP_SIZE)
        {
            voltage = cmadd(s_input[a], s_weight[warp][a], voltage);
        }

		s_inter[warp][warp_idx] = voltage;

		warp_reduce_v2p(
            &s_output[warp][tscrunch_cnt / INTERVAL_CB],
            &s_inter[warp][0], warp_idx);

        tscrunch_cnt++;

		__syncthreads();
        if(tscrunch_cnt / (INTERVAL_CB * WARP_SIZE) == 1)
        {
        	odata[odata_glob_idx + warp_idx] = s_output[warp][warp_idx];
			odata_glob_idx += WARP_SIZE;
        	tscrunch_cnt = 0;
		}
    }

    if(warp_idx < (N_TIMESTAMPS_CB / INTERVAL_CB) % WARP_SIZE && (N_TIMESTAMPS_CB / INTERVAL_CB) < WARP_SIZE)
    {
        odata[odata_glob_idx + warp_idx] = s_output[warp][warp_idx];
    }
}


template<typename T=float2, typename U=float>
__global__ void coherent_bf_power(const float2 *idata, float *odata, const float2 *weight)
{
}




/** Optimzed approach **/

template<typename T=float2, typename U=float>__global__
void bf_tfap_power(const float2 *idata, float *odata, const float2* weights, const bf_config_t conf)
{
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;      // Time dimension (T)
	int bidy = blockIdx.y;      // Channel dimension (F)

	const int n_elements = conf.n_antenna * conf.n_pol;
	const int n_timestamp = SHARED_IDATA / n_elements;	// Number of timestamps loaded into shared memory
	const int n_timestamp_iter = NTHREAD / n_elements;	// Number of timestamps loaded in one iteration by all active threads

	// WARP grouping
	const int warp_idx = tidx % WARP_SIZE;
	const int warp = tidx / WARP_SIZE;
	const int n_warps_per_grp = WARP_SIZE / n_timestamp_iter;
	const int warp_grp = warp / n_warps_per_grp;	// Devide all warps into groups to load one timestamp by each group
	const int warp_grp_idx = warp_idx + (warp - n_warps_per_grp * warp_grp) * WARP_SIZE; // Index of thread within a warp group

	const int idata_offset = bidx * conf.interval * conf.n_channel * n_elements + bidy * n_elements;
	const int odata_offset = bidy * conf.n_samples / conf.interval + bidx;		//

	int idata_glob_idx;
	T voltage, weight;


	__shared__ float2 s_idata[SHARED_IDATA]; // Shared memory for input data

	extern __shared__ unsigned char s_mem[];
	U *s_odata = reinterpret_cast<U*>(&s_mem[0]); // Shared memory for power data
	T *s_intermediate = reinterpret_cast<T*>(&s_mem[conf.n_beam * sizeof(U)]); // Shared memory for intermediate results
	// T *s_weights = reinterpret_cast<T*>(&s_mem[conf.n_beam * sizeof(U) + NTHREAD*sizeof(T)]);


	/* IMPORTANT: s_odata has to be initialized to zero for each element in the array*/
	for(int b = tidx; b < conf.n_beam; b += NTHREAD)
		if(b < conf.n_beam) s_odata[b] = 0;


	for(int t = 0; t < conf.interval; t += n_timestamp)
	{
		for(int i = 0; i < n_timestamp; i+=n_timestamp_iter)
		{
			idata_glob_idx = (t + warp_grp + i) * conf.n_channel * n_elements + warp_grp_idx;
			s_idata[i/n_timestamp_iter * NTHREAD + tidx] = idata[idata_offset + idata_glob_idx];
		}

		__syncthreads();

		for(int b = warp; b < conf.n_beam; b += WARPS)
		{
			// for(int a = warp_idx; a < n_elements; a += WARP_SIZE)
				// local_weights[a / WARP_SIZE] = tex3D<T>(weights, a, bidy, b);
			for(int i = 0; i < n_timestamp; i++)
			{
				voltage = {0,0};
				for(int a = warp_idx; a < n_elements; a += WARP_SIZE)
				{
					weight = weights[b * conf.n_channel * n_elements + bidy * n_elements + a];
					// Complex multiplication: raw voltage * weight = voltage
					voltage = cmadd(s_idata[i * n_elements + a], weight, voltage);
				}
				// Every thread accumulated n_elements/WARP_SIZE
				// Load accumulated result to shared memory; Every thread has its own field, otherwise race condition may occur
				s_intermediate[tidx] = voltage;
				// Reduction
				warp_reduce_v2p(&s_odata[b], &s_intermediate[warp * WARP_SIZE], warp_idx);
			}
		}
	}

	for(int b = tidx; b < conf.n_beam; b += NTHREAD)
	{
		if(b < conf.n_beam)
		{
			const int odata_glob_idx = b * conf.n_channel * conf.n_samples / conf.interval;
			const float power = s_odata[b] / conf.interval;
			odata[odata_offset + odata_glob_idx] = s_odata[b] / conf.interval;
		}
	}
}


template<typename T=__half2, typename U=__half>__global__
void bf_tfap_power(const __half2 *idata, __half *odata, const __half2 *weights, const bf_config_t conf)
{}





/** Optimized approach using texture memory **/

template<typename T=float2, typename U=float>__global__
void bf_tfap_power(const float2 *idata, float *odata, const cudaTextureObject_t weights, const bf_config_t conf)
{
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;      // Time dimension (T)
	int bidy = blockIdx.y;      // Channel dimension (F)

	const int n_elements = conf.n_antenna * conf.n_pol;
	const int n_timestamp = SHARED_IDATA / n_elements;	// Number of timestamps loaded into shared memory
	const int n_timestamp_iter = NTHREAD / n_elements;	// Number of timestamps loaded in one iteration by all active threads

	// WARP grouping
	const int warp_idx = tidx % WARP_SIZE;
	const int warp = tidx / WARP_SIZE;
	const int n_warps_per_grp = WARP_SIZE / n_timestamp_iter;
	const int warp_grp = warp / n_warps_per_grp;	// Devide all warps into groups to load one timestamp by each group
	const int warp_grp_idx = warp_idx + (warp - n_warps_per_grp * warp_grp) * WARP_SIZE; // Index of thread within a warp group

	const int idata_offset = bidx * conf.interval * conf.n_channel * n_elements + bidy * n_elements;
	const int odata_offset = bidy * conf.n_samples / conf.interval + bidx;		//

	int idata_glob_idx;
	T voltage, weight;


	__shared__ float2 s_idata[SHARED_IDATA]; // Shared memory for input data

	extern __shared__ unsigned char s_mem[];
	U *s_odata = reinterpret_cast<U*>(&s_mem[0]); // Shared memory for power data
	T *s_intermediate = reinterpret_cast<T*>(&s_mem[conf.n_beam * sizeof(U)]); // Shared memory for intermediate results


	/* IMPORTANT: s_odata has to be initialized to zero for each element in the array*/
	for(int b = tidx; b < conf.n_beam; b += NTHREAD)
		if(b < conf.n_beam) s_odata[b] = 0;


	for(int t = 0; t < conf.interval; t += n_timestamp)
	{
		for(int i = 0; i < n_timestamp; i+=n_timestamp_iter)
		{
			idata_glob_idx = (t + warp_grp + i) * conf.n_channel * n_elements + warp_grp_idx;
			s_idata[i/n_timestamp_iter * NTHREAD + tidx] = idata[idata_offset + idata_glob_idx];
		}

		__syncthreads();

		for(int b = warp; b < conf.n_beam; b += WARPS)
		{
			// for(int a = warp_idx; a < n_elements; a += WARP_SIZE)
				// local_weights[a / WARP_SIZE] = tex3D<T>(weights, a, bidy, b);
			for(int i = 0; i < n_timestamp; i++)
			{
				voltage = {0,0};
				for(int a = warp_idx; a < n_elements; a += WARP_SIZE)
				{
					weight = tex3D<T>(weights, a, bidy, b);
					// Complex multiplication: raw voltage * weight = voltage
					voltage = cmadd(s_idata[i * n_elements + a], weight, voltage);
				}
				// Every thread accumulated n_elements/WARP_SIZE
				// Load accumulated result to shared memory; Every thread has its own field, otherwise race condition may occur
				s_intermediate[tidx] = voltage;
				// Reduction
				warp_reduce_v2p(&s_odata[b], &s_intermediate[warp * WARP_SIZE], warp_idx);
			}
		}
	}

	for(int b = tidx; b < conf.n_beam; b += NTHREAD)
	{
		if(b < conf.n_beam)
		{
			const int odata_glob_idx = b * conf.n_channel * conf.n_samples / conf.interval;
			const float power = s_odata[b] / conf.interval;
			odata[odata_offset + odata_glob_idx] = s_odata[b] / conf.interval;
		}
	}
}


template<typename T=__half2, typename U=__half>__global__
void bf_tfap_power(const __half2 *idata, __half *odata, const cudaTextureObject_t weights, const bf_config_t conf)
{}


	// template<typename T=float2, typename U=float>__global__
	// void bf_tfap_power(const float2 *idata, float *odata, const float2* weights, const bf_config_t conf)
	// {
	// 	int tidx = threadIdx.x;
	// 	int bidx = blockIdx.x;      // Time dimension (T)
	// 	int bidy = blockIdx.y;      // Channel dimension (F)
	//
	// 	const int n_elements = conf.n_antenna * conf.n_pol;
	// 	const int n_timestamp = SHARED_IDATA / n_elements;	// Number of timestamps loaded into shared memory
	//
	// 	const int idata_offset = bidx * conf.interval * conf.n_channel * n_elements + bidy * n_elements;
	// 	const int odata_offset = bidy * conf.n_samples / conf.interval + bidx;		//
	//
	// 	int idata_glob_idx;
	//
	//
	// 	__shared__ float2 s_idata[SHARED_IDATA]; // Shared memory for input data
	// 	__shared__ float2 s_intermediate[WARPS][WARP_SIZE];
	//
	// 	extern __shared__ unsigned char s_mem[];
	// 	U *s_odata = reinterpret_cast<U*>(&s_mem[0]); // Shared memory for power data intermediate results
	// 	T *s_weights = reinterpret_cast<T*>(&s_mem[conf.n_beam * sizeof(U)]);
	//
	//
	// 	/* IMPORTANT: s_odata has to be initialized to zero for each element in the array*/
	// 	for(int b = tidx; b < conf.n_beam; b += NTHREAD)
	// 		s_odata[b] = 0;
	//
	// 	for(int t = 0; t < conf.interval; t += n_timestamp)
	// 	{
	// 		s_intermediate[warp][warp_idx] = {.0, .0};
	// 		for(int i = 0; i < n_timestamp; i++)
	// 		{
	// 			idata_glob_idx = (t + i) * conf.n_channel * n_elements + tidx;
	// 			s_idata[i * NTHREAD + tidx] = idata[idata_offset + idata_glob_idx];
	// 		}
	// 		for(int b = 0; b < conf.n_beam; b++)
	// 		{
	// 			for(int a = tidx; a < n_elements; a += NTHREAD)
	// 				s_weights[a] = weights[b * conf.n_channel * n_elements + bidy * n_elements + a];
	//
	// 			__syncthreads();
	//
	// 			for(int i = warp; i < n_timestamp; i+=WARPS)
	// 			{
	// 				// Complex multiplication: raw voltage * weight = voltage
	// 				for(int k = warp_idx; k < n_elements; k += WARP_SIZE)
	// 				{
	// 					s_intermediate[warp][warp_idx] = cmadd(s_idata[i * n_elements + k],  s_weights[k], s_intermediate[warp][warp_idx]);
	// 				}
	// 				// if(tidx == 0 && bidx == 0 && bidy == 0)
	// 				// 	printf("Val: %f + %f\n", )
	// 				// Every thread accumulated n_elements/WARP_SIZE
	// 				// Load accumulated result to shared memory; Every thread has its own field, otherwise race condition may occur
	// 				// Reduction
	// 				warp_reduce_v2p(&s_odata[b], s_intermediate[warp], warp_idx);
	// 			}
	// 		}
	// 	}
	//
	// 	for(int b = tidx; b < conf.n_beam; b += NTHREAD)
	// 	{
	// 		const int odata_glob_idx = b * conf.n_channel * conf.n_samples / conf.interval;
	// 		const float power = s_odata[b] / conf.interval;
	// 		odata[odata_offset + odata_glob_idx] = s_odata[b] / conf.interval;
	// 	}
	// }


/********************/
/** Naive approach **/
/********************/

template<typename T=float2, typename U=float>__global__
void simple_bf_tfap_power(const float2 *idata, float *odata, const float2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
	float2 acc{.0,.0};

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int i = 0; i < conf.n_antenna; i++)
		{
			acc.x = 0; acc.y = 0;
			for(int k = 0; k < conf.n_pol; k++)
			{
				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
		}
	}
}

template<typename T=__half2, typename U=__half>__global__
void simple_bf_tfap_power(const __half2 *idata, __half *odata, const __half2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
	__half2 acc(0,0);

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int i = 0; i < conf.n_antenna; i++)
		{
			acc.x = 0; acc.y = 0;
			for(int k = 0; k < conf.n_pol; k++)
			{
				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
		}
	}
}




}
}
}

#endif
