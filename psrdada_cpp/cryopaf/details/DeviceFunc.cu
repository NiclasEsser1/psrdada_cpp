
#include <hip/hip_runtime.h>
#ifdef CUKERNELS_CUH_


/** UTILS **/
__device__ __half2 __hCmul2(__half2 a, __half2 b)
{
		const __half r = a.x * b.x - a.y * b.y;
		const __half i = a.x * b.y + a.y * b.x;

		__half2 val; val.x = r; val.y = i;
		return val;
}

template<typename T>
__host__ __device__ T cmadd(T a, T b, T c)
{
	T val;
	val.x = a.x * b.x - a.y * b.y + c.x;
	val.y = a.x * b.y + a.y * b.x + c.y;
	return val;
}

template<typename T>
__host__ __device__ T cadd(T a, T b)
{
	T val;
	val.x = a.x + b.x;
	val.y = a.y + b.y;
	return val;
}

// template<typename T, typename U>
// __device__ void warp_reduce_all_elements_v2p(T *s_odata, U *s_idata, int warp_idx, int samples){
// 	n = samples / WARP_SIZE;
// 	for(int i = 1; i <= n; i+=WARP_SIZE)
// 	{
// 		s_idata[warp_idx] = cadd(s_idata[warp_idx], s_idata[warp_idx + WARP_SIZE * i]);
// 	}
// 	warp_reduce_v2p(s_odata, s_idata, warp_idx);
// }
template<typename T, typename U>
__device__ void warp_reduce_v2p(T *s_odata, U *s_idata, int warp_idx){
    if(warp_idx < 16)
    {
		// printf("%f + %f =",s_idata[warp_idx].x,s_idata[warp_idx + 16].x);
		s_idata[warp_idx] = cadd(s_idata[warp_idx], s_idata[warp_idx + 16]);
		// printf("%f\n",s_idata[warp_idx].x);
	}
	__syncthreads();
    if(warp_idx < 8)
	{
		// printf("%f + %f =",s_idata[warp_idx].x,s_idata[warp_idx + 8].x);
		s_idata[warp_idx] = cadd(s_idata[warp_idx], s_idata[warp_idx + 8]);
		// printf("%f\n",s_idata[warp_idx].x);
	}
	__syncthreads();
	if(warp_idx < 4)
	{
		// printf("%f + %f =",s_idata[warp_idx].x,s_idata[warp_idx + 4].x);
		s_idata[warp_idx] = cadd(s_idata[warp_idx], s_idata[warp_idx + 4]);
		// printf("%f\n",s_idata[warp_idx].x);
	}
	__syncthreads();
    if(warp_idx < 2)
	{
		// printf("%f + %f =",s_idata[warp_idx].x,s_idata[warp_idx + 2].x);
		s_idata[warp_idx] = cadd(s_idata[warp_idx], s_idata[warp_idx + 2]);
		// printf("%f\n",s_idata[warp_idx].x);
	}
	__syncthreads();
    if(warp_idx < 1)
    {
		// printf("%f\n",s_idata[warp_idx].x);

		T x_power = s_idata[warp_idx].x * s_idata[warp_idx].x + s_idata[warp_idx].y * s_idata[warp_idx].y;
		T y_power = s_idata[warp_idx + 1].x * s_idata[warp_idx + 1].x + s_idata[warp_idx + 1].y * s_idata[warp_idx + 1].y;
		// printf("x = %f = %f * %f\ny = %f = %f * %f\n",x_power, s_idata[warp_idx].x, s_idata[warp_idx].x, y_power,s_idata[warp_idx + 1].x,s_idata[warp_idx + 1].x);
    	s_odata[0] += x_power + y_power;
    }
	__syncthreads();

}
//
// /** STOKES I Beamformer **/
//
// /**
// *   @brief      coherent beamform kernel
//
// *   @params     idata, raw input voltages
// *   @params     weight, beamweights
// *   @params     odata, beamformed power data
// *
// *   @ detail    Every block computes 4 beams of 1 channel for all timestamps of a batched block
// */
// template<typename T=__half2, typename U=__half>
// __global__ void coherent_bf_power(const __half2 *idata, __half *odata, const __half2 *weight)
// {
//     const int tidx = threadIdx.x;
//     const int bidx = blockIdx.x; // Beams
//     const int bidy = blockIdx.y; // Channels
//     const int warp_idx = tidx % WARP_SIZE;
//     const int warp = tidx / WARP_SIZE;
//
//
//     int tscrunch_cnt = 0;
//
//     const int idata_glob_offset = bidy * N_ELEMENTS;
//     const int weight_glob_offset = bidx * WARPS * N_CHANNEL * N_ELEMENTS + bidy * N_ELEMENTS;
//     int odata_glob_idx = (bidx * WARPS + warp) * N_CHANNEL * N_OUTPUT_TIMESTAMPS + bidy * N_OUTPUT_TIMESTAMPS;
//
//     __shared__ __half2 s_input[N_ELEMENTS];
//     __shared__ __half2 s_weight[WARPS][N_ELEMENTS];
//     __shared__ __half2 s_inter[WARPS][WARP_SIZE];
//     __shared__ __half s_output[WARPS][WARP_SIZE];
//
//
//     for(int b = 0; b < WARPS; b++)
//     {
//         int weight_glob_idx = weight_glob_offset + b * N_CHANNEL * N_ELEMENTS;
//         for(int a = tidx; a < N_ELEMENTS; a+=NTHREAD)
//         {
//             s_weight[b][a] = weight[weight_glob_idx + a];
//         }
//     }
//
//     for(int t = 0; t < N_SAMPLES; t++)
//     {
//         int idata_glob_idx = idata_glob_offset + t * N_ELEMENTS * N_CHANNEL;
//         for(int a = tidx; a < N_ELEMENTS; a+=NTHREAD)
//         {
//             // All threads loading raw voltages
//             s_input[a] = idata[idata_glob_idx + a];
//         }
// 		__half2 voltage = {0,0};
//         __syncthreads();
//         // Each warp processes complex mulitplications for its beam
//         for(int a = warp_idx; a < N_ELEMENTS; a+=WARP_SIZE)
//         {
//             const __half2 voltage_inter = __hCmul2(s_input[a], s_weight[warp][a]);
// 			voltage.x += voltage_inter.x; voltage.y += voltage_inter.y;
//         }
//
// 		s_inter[warp][warp_idx] = voltage;
//
// 		warp_reduce_v2p(
//             &s_output[warp][tscrunch_cnt / INTERVAL],
//             s_inter[warp], warp_idx);
//
//         tscrunch_cnt++;
//
// 		__syncthreads();
//         if(tscrunch_cnt / (INTERVAL * WARP_SIZE) == 1)
//         {
//         	odata[odata_glob_idx + warp_idx] = s_output[warp][warp_idx];
// 			odata_glob_idx += WARP_SIZE;
//         	tscrunch_cnt = 0;
// 		}
//     }
//
//     if(warp_idx < (N_SAMPLES / INTERVAL) % WARP_SIZE && (N_SAMPLES / INTERVAL) < WARP_SIZE)
//     {
//         odata[odata_glob_idx + warp_idx] = s_output[warp][warp_idx];
//     }
// }
//
//
// template<typename T=float2, typename U=float>
// __global__ void coherent_bf_power(const float2 *idata, float *odata, const float2 *weight){}
//
//
// template<typename T=float2, typename U=float>__global__
// void simple_bf_tafpt_power(const float2 *idata, float *odata, const float2 *weights, const bf_config_t conf)
// {
// 	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
// 	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
// 	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
// 	float2 acc{.0,.0};
//
// 	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
// 	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
// 	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
//
// 	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
// 	{
// 		for(int i = 0; i < conf.n_antenna; i++)
// 		{
// 			acc.x = 0; acc.y = 0;
// 			for(int k = 0; k < conf.n_pol; k++)
// 			{
// 				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
// 					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
// 			}
// 			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
// 		}
// 	}
// }
//
// template<typename T=__half2, typename U=__half>__global__
// void simple_bf_tafpt_power(const __half2 *idata, __half *odata, const __half2 *weights, const bf_config_t conf)
// {
// 	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
// 	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
// 	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
// 	__half2 acc(0,0);
//
// 	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
// 	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
// 	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
//
// 	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
// 	{
// 		for(int i = 0; i < conf.n_antenna; i++)
// 		{
// 			acc.x = 0; acc.y = 0;
// 			for(int k = 0; k < conf.n_pol; k++)
// 			{
// 				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
// 					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
// 			}
// 			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
// 		}
// 	}
// }
//
//
// template<typename T=float2, typename U=float>__global__
// void bf_tafpt_power(const float2 *idata, float *odata, const cudaTextureObject_t weights, const bf_config_t conf)
// {
// 	int tidx = threadIdx.x;
// 	int bidx = blockIdx.x;      // Time dimension (T)
// 	int bidy = blockIdx.y;      // Channel dimension (F)
//
// 	const int n_elements = conf.n_antenna * conf.n_pol;
// 	const int n_timestamp = SHARED_IDATA / n_elements;	// Number of timestamps loaded into shared memory
// 	const int n_timestamp_iter = NTHREAD / n_elements;	// Number of timestamps loaded in one iteration by all active threads
//
// 	// WARP grouping
// 	const int warp_idx = tidx % WARP_SIZE;
// 	const int warp = tidx / WARP_SIZE;
// 	const int n_warps_per_grp = WARP_SIZE / n_timestamp_iter;
// 	const int warp_grp = warp / n_warps_per_grp;	// Devide all warps into groups to load one timestamp by each group
// 	const int warp_grp_idx = warp_idx + (warp - n_warps_per_grp * warp_grp) * WARP_SIZE; // Index of thread within a warp group
//
// 	const int idata_offset = bidx * conf.interval * conf.n_channel * n_elements + bidy * n_elements;
// 	const int odata_offset = bidy * conf.n_samples / conf.interval + bidx;		//
//
// 	int idata_glob_idx;
// 	float2 voltage, weight;
// 	float power;
//
//
// 	__shared__ float2 s_idata[SHARED_IDATA]; // Shared memory for input data
//
// 	extern __shared__ float s_mem[];
// 	float *s_odata = &s_mem[0]; // Shared memory for output data
// 	float *s_intermediate = &s_mem[conf.n_beam]; // Shared memory for intermediate results
//
//
// 	/* IMPORTANT: s_odata has to be initialized to zero for each element in the array*/
// 	for(int b = 0; b < conf.n_beam; b += NTHREAD)
// 	{
// 		if(b + tidx < conf.n_beam)
// 			s_odata[b + tidx] = 0;
// 	}
//
//
// 	for(int t = 0; t < conf.interval; t += n_timestamp)
// 	{
// 		for(int i = 0; i < n_timestamp; i+=n_timestamp_iter)
// 		{
// 			idata_glob_idx = (t + warp_grp + i) * conf.n_channel * n_elements + warp_grp_idx;
// 			s_idata[i/n_timestamp_iter * NTHREAD + tidx] = idata[idata_offset + idata_glob_idx];
// 		}
//
// 		__syncthreads();
//
// 		for(int b = 0; b < conf.n_beam; b += WARPS)
// 		{
// 			power = 0;
// 			for(int a = 0; a < n_elements; a += WARP_SIZE)
// 			{
// 					weight = tex3D<float2>(weights, (a + warp_idx), bidy, warp + b);
//
// 					for(int i = 0; i < n_timestamp; i++)
// 					{
// 						// Complex multiplication: raw voltage * weight = voltage
// 						voltage = cuCmulf(s_idata[i * n_elements + (a + warp_idx)], weight);
//
// 						// Cacluate (real) power; Square and root cancel each other out
// 						power += voltage.x * voltage.x + voltage.y * voltage.y;
// 					}
//
// 			}
// 			// Every thread accumulated polarizations + n_antenna/WARP_SIZE
// 			// Load accumulated result to shared memory; Every thread has its own field, otherwise race condition may occur
// 			s_intermediate[tidx] = power;
//
// 			// Reduction
// 			int i = WARP_SIZE/2;
// 			while(i != 0)
// 			{
// 				if(warp_idx < i)
// 					s_intermediate[tidx] += s_intermediate[tidx + i];
// 				i /= 2;
// 			}
//
//
// 			// After reduction the first warp adds intermediate results to dedicated shared output memory
// 			// 31/32 are idled :-(
// 			__syncthreads();
// 			if(warp_idx == 0)
// 				s_odata[b + warp] += s_intermediate[tidx];
// 		}
// 	}
//
// 	for(int b = 0; b < conf.n_beam; b += NTHREAD)
// 	{
// 		if(b + tidx < conf.n_beam)
// 		{
// 			const int odata_glob_idx = (b + tidx) * conf.n_channel * conf.n_samples / conf.interval;
// 			odata[odata_offset + odata_glob_idx] = s_odata[b + tidx] / conf.interval;
// 		}
// 	}
// }
//
//
// template<typename T=__half2, typename U=__half>__global__
// void bf_tafpt_power(const __half2 *idata, __half *odata, const cudaTextureObject_t weights, const bf_config_t conf)
// {}
//
//
//
//
//
// /** Voltage Beamformer **/
//
//
// template<typename T=float2>__global__
// void simple_bf_tafpt_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
// {
// 	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
// 	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
// 	int tidz = blockIdx.z * blockDim.z;									// Channel dimension
//
// 	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
// 	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
// 	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
//
// 	float2 acc;
//
// 	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
// 	{
// 		for(int k = 0; k < conf.n_pol; k++)
// 		{
// 			acc = {0,0};
// 			for(int i = 0; i < conf.n_antenna; i++)
// 			{
// 				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
// 					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
// 			}
// 			odata[out_offset + k] = acc;
// 		}
// 	}
// }
//
// template<typename T=__half2>__global__
// void simple_bf_tafpt_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
// {
// 	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
// 	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
// 	int tidz = blockIdx.z * blockDim.z;									// Channel dimension
//
// 	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
// 	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
// 	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
//
// 	__half2 acc;
//
// 	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
// 	{
// 		for(int k = 0; k < conf.n_pol; k++)
// 		{
// 			acc = {0,0};
// 			for(int i = 0; i < conf.n_antenna; i++)
// 			{
// 				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
// 						weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
// 			}
// 			odata[out_offset + k] = acc;
// 		}
// 	}
// }
//
//
// template<typename T=float2>__global__
// void bf_tfpat_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
// {
// 	// Grid layout: x = time; y = beam; z = channel
// 	// Block layout: A block consist of NTHREADS and WARPS. Every warp
// 	// calculates both polarisations of one beam for one channel at one given time step.
// 	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
// 	// Data products are as follow (glob mem):
// 	// 		idata: TFAP(t)
// 	//		odata: BFT
// 	//		weights: BFAP
// 	// constraints:
// 	//	- n_antenna must be a multiple of WARP_SIZE 32
// 	//	- n_samples must be a multiple of WARP_SIZE 32
//
// 	int tidx = threadIdx.x;
// 	int bidx = blockIdx.x;      // Time dimension (T)
// 	int bidy = blockIdx.y;      // Beam dimension (B)
// 	int bidz = blockIdx.z;      // Channel dimension (F)
//
// 	int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
// 	int warp = tidx / WARP_SIZE; // Calculate the current warp
// 	int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index
//
// 	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
// 	int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
// 	int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
// 	int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
// 			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations
//
// 	// To enable higher throughput and more efficient data transfer, shared memory
// 	// is required. The size in bytes of shared memory is calculated as follows:
// 	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
// 	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
// 	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
// 	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
// 	extern __shared__ float2 shared_mem_fp32[];	// dynamically allocated
// 	float2* shared_idata = (&shared_mem_fp32[0]);	// idata space comes first
// 	float2* shared_weights = (&shared_mem_fp32[n_elements * WARPS]);	// weight space with idata space as offset
//
// 	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuComplex).
// 	float2 __shared__ shared_odata[NTHREAD];
//
// 	shared_odata[tidx] = {0,0};	// intialize output with zeros
//
// 	float2 acc = {0,0}; // local register for storing intermediate results
//
// 	// Load idata and weights into shared memory for every warp
// #pragma unroll
// 	for(int i = 0; i < n_elements; i+=WARP_SIZE)
// 	{
// 		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
// 		// Here each thread within a warp accesses adjacent samples!
// 		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];
//
// 		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
// 		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
// 		// However, this approach prevents race conditions.
// 		if(warp == 0)
// 			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
// 	}
//
// 	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.
//
// 	// Iterate across all elements.
// 	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
// 	// 		FLOP/thread = n * (6+2)
// #pragma unroll
// 	for(int i = 0; i < n_elements; i+=WARP_SIZE)
// 	{
// 		acc = cuCaddf(acc, cuCmulf(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx]));
// 	}
// 	shared_odata[tidx] = acc;
//
// 	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.
//
// 	// Since odata contains NTHREAD samples which have not been combined to WARPS time steps a reduction is required.
// 	int i = WARP_SIZE / 2;
// 	// This reduction may no be very efficient since many threads within a warp are idled
// #pragma unroll
// 	while(i != conf.n_pol - 1)
// 	{
// 		if(warp_idx < i)
// 			shared_odata[tidx] = cuCaddf(shared_odata[tidx], shared_odata[tidx + i]);
// 		__syncthreads();
// 		i /= 2;
// 	}
// 	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
// 	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
// 	// samples are transfered back to the global memory for each block.
// 	if(warp_idx < conf.n_pol)
// 	{
// 		// TODO: In case of integer inputs, conversion is implemented here!!!
// 		odata[output_glob_idx] = shared_odata[tidx];	// Polarisation 0 and 1
// 	}
//
// }
//
//
// template<typename T=__half2>__global__
// void bf_tfpat_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
// {
//
// 	// Grid layout: x = time; y = beam; z = channel
// 	// Block layout: A block consist of 1024 threads and 32 warps. Every warp
// 	// calculates both polarisations of one beam for one channel at one given time step.
// 	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
// 	// Data products are as follow (glob mem):
// 	// 		idata: TFAP(t)
// 	//		odata: BFT
// 	//		weights: BFAP
// 	// constraints:
// 	//	- n_antenna must be a multiple of WARP_SIZE 32
// 	//	- n_samples must be a multiple of WARP_SIZE 32
//
// 	const int tidx = threadIdx.x;
// 	const int bidx = blockIdx.x;      // Time dimension (T)
// 	const int bidy = blockIdx.y;      // Beam dimension (B)
// 	const int bidz = blockIdx.z;      // Channel dimension (F)
// #if __CUDA_ARCH__ >= 530
// 	const int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
// 	const int warp = tidx / WARP_SIZE; // Calculate the current warp
// 	const int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index
//
// 	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
// 	const int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
// 	const int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
// 	const int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
// 			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations
//
// 	// To enable higher throughput and more efficient data transfer, shared memory
// 	// is required. The size in bytes of shared memory is calculated as follows:
// 	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
// 	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
// 	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
// 	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
// 	extern __shared__ __half2 shared_mem_fp16[];	// dynamically allocated
// 	__half2* shared_idata = (&shared_mem_fp16[0]);	// idata space comes first
// 	__half2* shared_weights = (&shared_mem_fp16[n_elements * WARPS]);	// weight space with idata space as offset
//
// 	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuCOmplex).
// 	__half2 __shared__ shared_odata[NTHREAD];
//
// 	shared_odata[tidx] = {0,0};	// intialize output with zeros
//
// 	__half2 acc = {0,0}; // local register for storing intermediate results
//
// 	// Load idata and weights into shared memory for every warp
// #pragma unroll
// 	for(int i = 0; i < n_elements; i+=WARP_SIZE)
// 	{
// 		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
// 		// Here each thread within a warp accesses adjacent samples!
// 		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];
//
// 		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
// 		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
// 		// However, this approach prevents race conditions.
// 		if(warp == 0)
// 			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
// 	}
//
// 	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.
//
// 	// Iterate across all elements.
// 	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
// 	// 		FLOP/thread = n * (6+2)
// #pragma unroll
// 	for(int i = 0; i < n_elements; i+=WARP_SIZE)
// 	{
// 		acc = __hadd2(acc, (__hCmul2(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx])));
// 	}
// 	shared_odata[tidx] = (acc);
//
// 	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.
//
// 	// Since odata contains 1024 samples which have not been combined to 32 time steps a reduction is required.
// 	int i = WARP_SIZE / 2;
// 	// This reduction may no be very efficient since many threads within a warp are idled
// #pragma unroll
// 	while(i != conf.n_pol - 1)
// 	{
// 		if(warp_idx < i)
// 			shared_odata[tidx] = __hadd2(shared_odata[tidx], shared_odata[tidx + i]);
// 		__syncthreads();
// 		i /= 2;
// 	}
//
// 	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
// 	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
// 	// samples are transfered back to the global memory for each block.
// 	if(warp_idx < conf.n_pol)
// 	{
// 		// TODO: In case of integer inputs, conversion is implemented here!!!
// 		odata[output_glob_idx] = (shared_odata[tidx]);	// Polarisation 0 and 1
// 	}
//
// #else
// if(tidx == 0 && bidx==0 && bidy == 0 && bidz == 0)
// 	printf("Warning: CUDA architecture does not support half precisison. Beamforming not executed...\n");
// #endif
//
// }

#endif /* CUKERNELS_CUH_ */
