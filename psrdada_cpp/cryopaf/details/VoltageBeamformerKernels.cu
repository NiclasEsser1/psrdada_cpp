
#include <hip/hip_runtime.h>
#ifdef VOLTAGE_BEAMFORMER_CUH_

namespace psrdada_cpp{
namespace cryopaf{
namespace beamforming{


/** Voltage Beamformer **/
template<typename T=float2>__global__
void bf_tfpa_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
{
	// Grid layout: x = time; y = beam; z = channel
	// Block layout: A block consist of NTHREADS and WARPS. Every warp
	// calculates both polarisations of one beam for one channel at one given time step.
	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
	// Data products are as follow (glob mem):
	// 		idata: TFAP(t)
	//		odata: BFT
	//		weights: BFAP
	// constraints:
	//	- n_antenna must be a multiple of WARP_SIZE 32
	//	- n_samples must be a multiple of WARP_SIZE 32

	int tidx = threadIdx.x;
	int bidx = blockIdx.x;      // Time dimension (T)
	int bidy = blockIdx.y;      // Beam dimension (B)
	int bidz = blockIdx.z;      // Channel dimension (F)

	int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
	int warp = tidx / WARP_SIZE; // Calculate the current warp
	int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index

	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
	int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
	int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
	int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations

	// To enable higher throughput and more efficient data transfer, shared memory
	// is required. The size in bytes of shared memory is calculated as follows:
	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
	extern __shared__ float2 shared_mem_fp32[];	// dynamically allocated
	float2* shared_idata = (&shared_mem_fp32[0]);	// idata space comes first
	float2* shared_weights = (&shared_mem_fp32[n_elements * WARPS]);	// weight space with idata space as offset

	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuComplex).
	float2 __shared__ shared_odata[NTHREAD];

	shared_odata[tidx] = {0,0};	// intialize output with zeros

	float2 acc = {0,0}; // local register for storing intermediate results

	// Load idata and weights into shared memory for every warp
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
		// Here each thread within a warp accesses adjacent samples!
		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];

		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
		// However, this approach prevents race conditions.
		if(warp == 0)
			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
	}

	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.

	// Iterate across all elements.
	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
	// 		FLOP/thread = n * (6+2)
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		acc = cuCaddf(acc, cuCmulf(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx]));
	}
	shared_odata[tidx] = acc;

	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.

	// Since odata contains NTHREAD samples which have not been combined to WARPS time steps a reduction is required.
	int i = WARP_SIZE / 2;
	// This reduction may no be very efficient since many threads within a warp are idled
#pragma unroll
	while(i != conf.n_pol - 1)
	{
		if(warp_idx < i)
			shared_odata[tidx] = cuCaddf(shared_odata[tidx], shared_odata[tidx + i]);
		__syncthreads();
		i /= 2;
	}
	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
	// samples are transfered back to the global memory for each block.
	if(warp_idx < conf.n_pol)
	{
		// TODO: In case of integer inputs, conversion is implemented here!!!
		odata[output_glob_idx] = shared_odata[tidx];	// Polarisation 0 and 1
	}

}


template<typename T=__half2>__global__
void bf_tfpa_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
{
	// Grid layout: x = time; y = beam; z = channel
	// Block layout: A block consist of 1024 threads and 32 warps. Every warp
	// calculates both polarisations of one beam for one channel at one given time step.
	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
	// Data products are as follow (glob mem):
	// 		idata: TFAP(t)
	//		odata: BFT
	//		weights: BFAP
	// constraints:
	//	- n_antenna must be a multiple of WARP_SIZE 32
	//	- n_samples must be a multiple of WARP_SIZE 32

	const int tidx = threadIdx.x;
	const int bidx = blockIdx.x;      // Time dimension (T)
	const int bidy = blockIdx.y;      // Beam dimension (B)
	const int bidz = blockIdx.z;      // Channel dimension (F)
#if __CUDA_ARCH__ >= 530
	const int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
	const int warp = tidx / WARP_SIZE; // Calculate the current warp
	const int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index

	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
	const int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
	const int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
	const int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations

	// To enable higher throughput and more efficient data transfer, shared memory
	// is required. The size in bytes of shared memory is calculated as follows:
	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
	extern __shared__ __half2 shared_mem_fp16[];	// dynamically allocated
	__half2* shared_idata = (&shared_mem_fp16[0]);	// idata space comes first
	__half2* shared_weights = (&shared_mem_fp16[n_elements * WARPS]);	// weight space with idata space as offset

	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuCOmplex).
	__half2 __shared__ shared_odata[NTHREAD];

	shared_odata[tidx] = {0,0};	// intialize output with zeros

	__half2 acc = {0,0}; // local register for storing intermediate results

	// Load idata and weights into shared memory for every warp
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
		// Here each thread within a warp accesses adjacent samples!
		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];

		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
		// However, this approach prevents race conditions.
		if(warp == 0)
			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
	}

	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.

	// Iterate across all elements.
	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
	// 		FLOP/thread = n * (6+2)
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		acc = __hadd2(acc, (__hCmul2(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx])));
	}
	shared_odata[tidx] = (acc);

	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.

	// Since odata contains 1024 samples which have not been combined to 32 time steps a reduction is required.
	int i = WARP_SIZE / 2;
	// This reduction may no be very efficient since many threads within a warp are idled
#pragma unroll
	while(i != conf.n_pol - 1)
	{
		if(warp_idx < i)
			shared_odata[tidx] = __hadd2(shared_odata[tidx], shared_odata[tidx + i]);
		__syncthreads();
		i /= 2;
	}

	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
	// samples are transfered back to the global memory for each block.
	if(warp_idx < conf.n_pol)
	{
		// TODO: In case of integer inputs, conversion is implemented here!!!
		odata[output_glob_idx] = (shared_odata[tidx]);	// Polarisation 0 and 1
	}

#else
if(tidx == 0 && bidx==0 && bidy == 0 && bidz == 0)
	printf("Warning: CUDA architecture does not support half precisison. Beamforming not executed...\n");
#endif

}

template<typename T=float2>__global__
void simple_bf_tafp_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
	int tidz = blockIdx.z * blockDim.z;									// Channel dimension

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	float2 acc;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int k = 0; k < conf.n_pol; k++)
		{
			acc = {0,0};
			for(int i = 0; i < conf.n_antenna; i++)
			{
				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + k] = acc;
		}
	}
}

template<typename T=__half2>__global__
void simple_bf_tafp_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
	int tidz = blockIdx.z * blockDim.z;									// Channel dimension

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	__half2 acc;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int k = 0; k < conf.n_pol; k++)
		{
			acc = {0,0};
			for(int i = 0; i < conf.n_antenna; i++)
			{
				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
						weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + k] = acc;
		}
	}
}


}
}
}

#endif
