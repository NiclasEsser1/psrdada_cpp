
#include <hip/hip_runtime.h>
#ifdef UNPACKER_CUH

namespace psrdada_cpp {
namespace cryopaf{


__device__ __forceinline__ uint64_t swap64(uint64_t x)
{
    uint64_t result;
    uint2 t;
    asm("mov.b64 {%0,%1},%2; \n\t"
        : "=r"(t.x), "=r"(t.y) : "l"(x));
    t.x = __byte_perm(t.x, 0, 0x0123);
    t.y = __byte_perm(t.y, 0, 0x0123);
    asm("mov.b64 %0,{%1,%2}; \n\t"
        : "=l"(result) : "r"(t.y), "r"(t.x));
    return result;
}


template<typename T>__global__
void unpack_codif_to_fpte(uint64_t const* __restrict__ idata, T* __restrict__ odata)
{
    int time = threadIdx.x + blockIdx.x * blockDim.x; // Time
    int elem = threadIdx.y + blockIdx.y * blockDim.y; // Elements
    int freq = threadIdx.z + blockIdx.z * blockDim.z; // Frequency
    int chan = blockDim.z * gridDim.z;

    int time_in = blockIdx.x * blockDim.x * gridDim.y * chan + threadIdx.x * chan;
    int freq_in = freq;
    int elem_in = elem * NSAMP_DF * chan ;

    int freq_out = freq * NPOL_SAMP * gridDim.x * blockDim.x * gridDim.y;
    int time_out = time * gridDim.y;

    int in_idx = time_in + freq_in + elem_in;
    int out_idx_x = freq_out + time_out + elem;
    int out_idx_y = freq_out + gridDim.x * blockDim.x * gridDim.y + time_out + elem;

    uint64_t tmp = swap64(idata[in_idx]);

    odata[out_idx_x].x = static_cast<decltype(T::x)>((tmp & 0x000000000000ffffLL));
    odata[out_idx_x].y = static_cast<decltype(T::y)>((tmp & 0x00000000ffff0000LL) >> 16);

    odata[out_idx_y].x = static_cast<decltype(T::x)>((tmp & 0x0000ffff00000000LL) >> 32);
    odata[out_idx_y].y = static_cast<decltype(T::y)>((tmp & 0xffff000000000000LL) >> 48);
}

template<typename U, typename T>__global__
void unpack_spead_ttfep_to_fpte(U const* __restrict__ idata, T* __restrict__ odata)
{
    int time = threadIdx.x; // Time
    int elem = blockIdx.y; // Elements
    int freq = blockIdx.z; // Frequency
    int heap_idx = blockIdx.x;

    int in_idx = heap_idx * NSAMP_PER_HEAP * gridDim.z * gridDim.y * NPOL_SAMP // Outer time axis
      + time * gridDim.z * gridDim.y * NPOL_SAMP // Inner time axis
      + freq * gridDim.y * NPOL_SAMP // Frequency axis
      + elem * NPOL_SAMP; // Element axis

    int out_idx_x = freq * NPOL_SAMP * gridDim.x * NSAMP_PER_HEAP * gridDim.y // Frequency axis
      + (time + blockIdx.x * blockDim.x) * gridDim.y
      + elem;
    int out_idx_y = freq * NPOL_SAMP * gridDim.x * NSAMP_PER_HEAP * gridDim.y // Frequency axis
      + gridDim.x * NSAMP_PER_HEAP * gridDim.y
      + (time + blockIdx.x * blockDim.x) * gridDim.y
      + elem;

    odata[out_idx_x].x = static_cast<decltype(T::x)>(idata[in_idx].x);
    odata[out_idx_x].y = static_cast<decltype(T::y)>(idata[in_idx].y);

    odata[out_idx_y].x = static_cast<decltype(T::x)>(idata[in_idx + 1].x);
    odata[out_idx_y].y = static_cast<decltype(T::y)>(idata[in_idx + 1].y);
}

// ######################################################
// NOTE: Kernels above are deprecated and not longer used
// ######################################################
/*
template<typename T>__global__
void unpack_codif_to_tfep(uint64_t const* __restrict__ idata, T* __restrict__ odata)
{

    int time = threadIdx.x + blockIdx.x * blockDim.x; // Time
    int elem = threadIdx.y + blockIdx.y * blockDim.y; // Elements
    int freq = threadIdx.z + blockIdx.z * blockDim.z; // Frequency
    int chan = blockDim.z * gridDim.z;

    int time_in = blockIdx.x * blockDim.x * gridDim.y * chan + threadIdx.x * chan;
    int freq_in = freq;
    int elem_in = elem * NSAMP_DF * chan ;

    int time_out = time * chan * gridDim.y * NPOL_SAMP;
    int freq_out = freq * gridDim.y * NPOL_SAMP;
    int elem_out = elem * NPOL_SAMP;

    int in_idx = time_in + freq_in + elem_in;
    int out_idx = time_out + freq_out + elem_out;

    uint64_t tmp = swap64(idata[in_idx]);

    odata[out_idx].x = static_cast<decltype(T::x)>((tmp & 0x000000000000ffffLL));
    odata[out_idx].y = static_cast<decltype(T::y)>((tmp & 0x00000000ffff0000LL) >> 16);

    odata[out_idx + 1].x = static_cast<decltype(T::x)>((tmp & 0x0000ffff00000000LL) >> 32);
    odata[out_idx + 1].y = static_cast<decltype(T::y)>((tmp & 0xffff000000000000LL) >> 48);
}
*/
}
}

#endif
