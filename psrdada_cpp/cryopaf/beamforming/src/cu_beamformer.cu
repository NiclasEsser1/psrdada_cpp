
#include <hip/hip_runtime.h>
// #include "psrdada_cpp/cryopaf/beamforming/cu_beamformer.cuh"

#ifdef CUDA_BEAMFORMER_CUH_

namespace psrdada_cpp{
namespace cryopaf{
namespace beamforming{


template<class T>
CudaBeamformer<T>::CudaBeamformer(bf_config_t *conf, int device_id)
	: _conf(conf), id(device_id)
{
	// std::cout << "Creating instance of CudaBeamformer" << std::endl;
	// Set device to use
	CUDA_ERROR_CHECK(cudaSetDevice(id))
	// Retrieve device properties
	CUDA_ERROR_CHECK(cudaGetDeviceProperties(&prop, id))
	// initialize beamformer enviroment
	init();

}


template<class T>
CudaBeamformer<T>::~CudaBeamformer()
{
	// std::cout << "Destroying instance of CudaBeamformer" << std::endl;
}


template<class T>
void CudaBeamformer<T>::kernel_layout()
{
	switch(_conf->bf_type){
		case SIMPLE_BF_TAFPT:
			grid_layout.x = (_conf->n_samples < NTHREAD) ? 1 : _conf->n_samples/NTHREAD;
			grid_layout.y = _conf->n_beam;
			grid_layout.z = _conf->n_channel;
			block_layout.x = NTHREAD; //(_conf->n_samples < NTHREAD) ? _conf->n_samples : NTHREAD;
			break;
		case BF_TFAP:
			// shared_mem_bytes = sizeof(T) * (_conf->n_antenna * _conf->n_pol * (WARPS + 1)); // TODO: This is not true for power /stokes I
			shared_mem_bytes = sizeof(T) * (_conf->n_antenna * _conf->n_pol * (WARPS + 1)); // TODO: This is not true for power /stokes I
			std::cout << "Required shared memory: " << std::to_string(shared_mem_bytes) << " Bytes" << std::endl;
			if(prop.sharedMemPerBlock < shared_mem_bytes + NTHREAD)
			{
				std::cout << "The requested size for shared memory per block exceeds the size provided by device "
					<< std::to_string(id) << std::endl
					<< "! Warning: Kernel will not get launched !" << std::endl;
					success = false;
					return;
			}else{
				success = true;
			}
			grid_layout.x = _conf->n_samples * WARP_SIZE / (NTHREAD);
			grid_layout.y = _conf->n_beam;
			grid_layout.z = _conf->n_channel;
			block_layout.x = NTHREAD;
			break;

		default:
			std::cout << "Beamform type not known..." << std::endl;
			break;
	}
}


template<class T>
void CudaBeamformer<T>::init(bf_config_t *conf)
{
	// If new configuration is passed
	if(conf){_conf = conf;}
	// Make kernel layout for GPU-Kernel
	kernel_layout();
	// if(success)
	// 	print_layout();
}


template<class T>
template<class U>
void CudaBeamformer<T>::process(
	const thrust::device_vector<T>& in,
	thrust::device_vector<U>& out,
	const thrust::device_vector<T>& weights,
	cudaStream_t stream)
{
	if(!success){return;}
	// Cast raw data pointer for passing to CUDA kernel
	const T *p_in = thrust::raw_pointer_cast(in.data());
	const T *p_weights = thrust::raw_pointer_cast(weights.data());
	U *p_out = thrust::raw_pointer_cast(out.data());

	if constexpr (!std::is_same<T, U>::value)
	{
		// Switch to desired CUDA kernel
		switch(_conf->bf_type)
		{
			case SIMPLE_BF_TAFPT:
			{
				std::cout << "Power beamformer (Stokes I): simple TFAPT" << std::endl;
				simple_bf_tafpt_power<<<grid_layout, block_layout>>>(p_in, p_out, p_weights, *_conf);
				break;
			}
			case BF_TFAP:
			{
				std::cout << "Power beamformer (Stokes I): optimized TFAPT" << std::endl;
				bf_tafpt_power<<<grid_layout, block_layout>>>(p_in, p_out, p_weights, *_conf);
				break;
			}
			default:
				std::cout << "Beamform type " << std::to_string(_conf->bf_type) << " not known..." << std::endl;
				break;
		}
	}
	else
	{
		// Switch to desired CUDA kernel
		switch(_conf->bf_type)
		{
			case SIMPLE_BF_TAFPT:
			{
				std::cout << "Voltage beamformer: simple TAFPT" << std::endl;
				simple_bf_tafpt_voltage<<<grid_layout, block_layout>>>(p_in, p_out, p_weights, *_conf);
				break;
			}
			case BF_TFAP:
			{
				std::cout << "Voltage beamformer: optimzed TFAPT" << std::endl;
					bf_tfpat_voltage<<<grid_layout, block_layout, shared_mem_bytes>>>(p_in, p_out, p_weights, *_conf);
				break;
			}

			default:
				std::cout << "Beamform type " << std::to_string(_conf->bf_type) << "not known..." << std::endl;
				break;
		}
	}
}


template<class T>
void CudaBeamformer<T>::print_layout()
{
	std::cout << " Kernel layout: " << std::endl
		<< " g.x = " << std::to_string(grid_layout.x) << std::endl
		<< " g.y = " << std::to_string(grid_layout.y) << std::endl
		<< " g.z = " << std::to_string(grid_layout.z) << std::endl
		<< " b.x = " << std::to_string(block_layout.x)<< std::endl
		<< " b.y = " << std::to_string(block_layout.y)<< std::endl
		<< " b.z = " << std::to_string(block_layout.z)<< std::endl;
}

} // namespace beamforming
} // namespace cryopaf
} // namespace psrdada_cpp

#endif
