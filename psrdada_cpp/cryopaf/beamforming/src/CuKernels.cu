
#include <hip/hip_runtime.h>
#ifdef CUKERNELS_CUH_


/** UTILS **/
__device__ __half2 __hCmul2(__half2 a, __half2 b)
{
		const __half r = a.x * b.x - a.y * b.y;
		const __half i = a.x * b.y + a.y * b.x;

		__half2 val; val.x = r; val.y = i;
		return val;
}


/** STOKES I Beamformer **/
template<typename T=float2, typename U=float>__global__
void simple_bf_tafpt_power(const float2 *idata, float *odata, const float2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
	float2 acc{.0,.0};

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int i = 0; i < conf.n_antenna; i++)
		{
			acc.x = 0; acc.y = 0;
			for(int k = 0; k < conf.n_pol; k++)
			{
				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
		}
	}
}

template<typename T=__half2, typename U=__half>__global__
void simple_bf_tafpt_power(const __half2 *idata, __half *odata, const __half2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
	__half2 acc(0,0);

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int i = 0; i < conf.n_antenna; i++)
		{
			acc.x = 0; acc.y = 0;
			for(int k = 0; k < conf.n_pol; k++)
			{
				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
		}
	}
}


template<typename T=float2, typename U=float>__global__
void bf_tafpt_power(const float2 *idata, float *odata, const cudaTextureObject_t weights, const bf_config_t conf)
{
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;      // Time dimension (T)
	int bidy = blockIdx.y;      // Channel dimension (F)

	const int n_elements = conf.n_antenna * conf.n_pol;
	const int n_timestamp = SHARED_IDATA / n_elements;	// Number of timestamps loaded into shared memory
	const int n_timestamp_iter = NTHREAD / n_elements;	// Number of timestamps loaded in one iteration by all active threads

	// WARP grouping
	const int warp_idx = tidx % WARP_SIZE;
	const int warp = tidx / WARP_SIZE;
	const int n_warps_per_grp = WARP_SIZE / n_timestamp_iter;
	const int warp_grp = warp / n_warps_per_grp;	// Devide all warps into groups to load one timestamp by each group
	const int warp_grp_idx = warp_idx + (warp - n_warps_per_grp * warp_grp) * WARP_SIZE; // Index of thread within a warp group

	const int idata_offset = bidx * conf.interval * conf.n_channel * n_elements + bidy * n_elements;
	const int odata_offset = bidy * conf.n_samples / conf.interval + bidx;		//

	int idata_glob_idx;
	float2 voltage, weight;
	float power;


	__shared__ float2 s_idata[SHARED_IDATA]; // Shared memory for input data

	extern __shared__ float s_mem[];
	float *s_odata = &s_mem[0]; // Shared memory for output data
	float *s_intermediate = &s_mem[conf.n_beam]; // Shared memory for intermediate results


	/* IMPORTANT: s_odata has to be initialized to zero for each element in the array*/
	for(int b = 0; b < conf.n_beam; b += NTHREAD)
	{
		if(b + tidx < conf.n_beam)
			s_odata[b + tidx] = 0;
	}


	for(int t = 0; t < conf.interval; t += n_timestamp)
	{
		for(int i = 0; i < n_timestamp; i+=n_timestamp_iter)
		{
			idata_glob_idx = (t + warp_grp + i) * conf.n_channel * n_elements + warp_grp_idx;
			s_idata[i/n_timestamp_iter * NTHREAD + tidx] = idata[idata_offset + idata_glob_idx];
		}

		__syncthreads();

		for(int b = 0; b < conf.n_beam; b += WARPS)
		{
			power = 0;
			for(int a = 0; a < n_elements; a += WARP_SIZE)
			{
					weight = tex3D<float2>(weights, (a + warp_idx), bidy, warp + b);

					for(int i = 0; i < n_timestamp; i++)
					{
						// Complex multiplication: raw voltage * weight = voltage
						voltage = cuCmulf(s_idata[i * n_elements + (a + warp_idx)], weight);

						// Cacluate (real) power; Square and root cancel each other out
						power += voltage.x * voltage.x + voltage.y * voltage.y;
					}

			}
			// Every thread accumulated polarizations + n_antenna/WARP_SIZE
			// Load accumulated result to shared memory; Every thread has its own field, otherwise race condition may occur
			s_intermediate[tidx] = power;

			// Reduction
			int i = WARP_SIZE/2;
			while(i != 0)
			{
				if(warp_idx < i)
					s_intermediate[tidx] += s_intermediate[tidx + i];
				i /= 2;
			}


			// After reduction the first warp adds intermediate results to dedicated shared output memory
			// 31/32 are idled :-(
			__syncthreads();
			if(warp_idx == 0)
				s_odata[b + warp] += s_intermediate[tidx];
		}
	}

	for(int b = 0; b < conf.n_beam; b += NTHREAD)
	{
		if(b + tidx < conf.n_beam)
		{
			const int odata_glob_idx = (b + tidx) * conf.n_channel * conf.n_samples / conf.interval;
			odata[odata_offset + odata_glob_idx] = s_odata[b + tidx] / conf.interval;
		}
	}
}


template<typename T=__half2, typename U=__half>__global__
void bf_tafpt_power(const __half2 *idata, __half *odata, const cudaTextureObject_t weights, const bf_config_t conf)
{
	// int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	// int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	// int tidz = blockIdx.z * blockDim.z;								// Antenna dimension



}


/** Voltage Beamformer **/


template<typename T=float2>__global__
void simple_bf_tafpt_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
	int tidz = blockIdx.z * blockDim.z;									// Channel dimension

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	float2 acc;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int k = 0; k < conf.n_pol; k++)
		{
			acc = {0,0};
			for(int i = 0; i < conf.n_antenna; i++)
			{
				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + k] = acc;
		}
	}
}

template<typename T=__half2>__global__
void simple_bf_tafpt_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
	int tidz = blockIdx.z * blockDim.z;									// Channel dimension

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	__half2 acc;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int k = 0; k < conf.n_pol; k++)
		{
			acc = {0,0};
			for(int i = 0; i < conf.n_antenna; i++)
			{
				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
						weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + k] = acc;
		}
	}
}


template<typename T=float2>__global__
void bf_tfpat_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
{
	// Grid layout: x = time; y = beam; z = channel
	// Block layout: A block consist of NTHREADS and WARPS. Every warp
	// calculates both polarisations of one beam for one channel at one given time step.
	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
	// Data products are as follow (glob mem):
	// 		idata: TFAP(t)
	//		odata: BFT
	//		weights: BFAP
	// constraints:
	//	- n_antenna must be a multiple of WARP_SIZE 32
	//	- n_samples must be a multiple of WARP_SIZE 32

	int tidx = threadIdx.x;
	int bidx = blockIdx.x;      // Time dimension (T)
	int bidy = blockIdx.y;      // Beam dimension (B)
	int bidz = blockIdx.z;      // Channel dimension (F)

	int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
	int warp = tidx / WARP_SIZE; // Calculate the current warp
	int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index

	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
	int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
	int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
	int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations

	// To enable higher throughput and more efficient data transfer, shared memory
	// is required. The size in bytes of shared memory is calculated as follows:
	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
	extern __shared__ float2 shared_mem_fp32[];	// dynamically allocated
	float2* shared_idata = (&shared_mem_fp32[0]);	// idata space comes first
	float2* shared_weights = (&shared_mem_fp32[n_elements * WARPS]);	// weight space with idata space as offset

	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuComplex).
	float2 __shared__ shared_odata[NTHREAD];

	shared_odata[tidx] = {0,0};	// intialize output with zeros

	float2 acc = {0,0}; // local register for storing intermediate results

	// Load idata and weights into shared memory for every warp
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
		// Here each thread within a warp accesses adjacent samples!
		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];

		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
		// However, this approach prevents race conditions.
		if(warp == 0)
			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
	}

	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.

	// Iterate across all elements.
	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
	// 		FLOP/thread = n * (6+2)
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		acc = cuCaddf(acc, cuCmulf(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx]));
	}
	shared_odata[tidx] = acc;

	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.

	// Since odata contains NTHREAD samples which have not been combined to WARPS time steps a reduction is required.
	int i = WARP_SIZE / 2;
	// This reduction may no be very efficient since many threads within a warp are idled
#pragma unroll
	while(i != conf.n_pol - 1)
	{
		if(warp_idx < i)
			shared_odata[tidx] = cuCaddf(shared_odata[tidx], shared_odata[tidx + i]);
		__syncthreads();
		i /= 2;
	}
	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
	// samples are transfered back to the global memory for each block.
	if(warp_idx < conf.n_pol)
	{
		// TODO: In case of integer inputs, conversion is implemented here!!!
		odata[output_glob_idx] = shared_odata[tidx];	// Polarisation 0 and 1
	}

}


template<typename T=__half2>__global__
void bf_tfpat_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
{

	// Grid layout: x = time; y = beam; z = channel
	// Block layout: A block consist of 1024 threads and 32 warps. Every warp
	// calculates both polarisations of one beam for one channel at one given time step.
	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
	// Data products are as follow (glob mem):
	// 		idata: TFAP(t)
	//		odata: BFT
	//		weights: BFAP
	// constraints:
	//	- n_antenna must be a multiple of WARP_SIZE 32
	//	- n_samples must be a multiple of WARP_SIZE 32

	const int tidx = threadIdx.x;
	const int bidx = blockIdx.x;      // Time dimension (T)
	const int bidy = blockIdx.y;      // Beam dimension (B)
	const int bidz = blockIdx.z;      // Channel dimension (F)
#if __CUDA_ARCH__ >= 530
	const int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
	const int warp = tidx / WARP_SIZE; // Calculate the current warp
	const int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index

	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
	const int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
	const int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
	const int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations

	// To enable higher throughput and more efficient data transfer, shared memory
	// is required. The size in bytes of shared memory is calculated as follows:
	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
	extern __shared__ __half2 shared_mem_fp16[];	// dynamically allocated
	__half2* shared_idata = (&shared_mem_fp16[0]);	// idata space comes first
	__half2* shared_weights = (&shared_mem_fp16[n_elements * WARPS]);	// weight space with idata space as offset

	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuCOmplex).
	__half2 __shared__ shared_odata[NTHREAD];

	shared_odata[tidx] = {0,0};	// intialize output with zeros

	__half2 acc = {0,0}; // local register for storing intermediate results

	// Load idata and weights into shared memory for every warp
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
		// Here each thread within a warp accesses adjacent samples!
		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];

		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
		// However, this approach prevents race conditions.
		if(warp == 0)
			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
	}

	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.

	// Iterate across all elements.
	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
	// 		FLOP/thread = n * (6+2)
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		acc = __hadd2(acc, (__hCmul2(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx])));
	}
	shared_odata[tidx] = (acc);

	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.

	// Since odata contains 1024 samples which have not been combined to 32 time steps a reduction is required.
	int i = WARP_SIZE / 2;
	// This reduction may no be very efficient since many threads within a warp are idled
#pragma unroll
	while(i != conf.n_pol - 1)
	{
		if(warp_idx < i)
			shared_odata[tidx] = __hadd2(shared_odata[tidx], shared_odata[tidx + i]);
		__syncthreads();
		i /= 2;
	}

	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
	// samples are transfered back to the global memory for each block.
	if(warp_idx < conf.n_pol)
	{
		// TODO: In case of integer inputs, conversion is implemented here!!!
		odata[output_glob_idx] = (shared_odata[tidx]);	// Polarisation 0 and 1
	}

#else
if(tidx == 0 && bidx==0 && bidy == 0 && bidz == 0)
	printf("Warning: CUDA architecture does not support half precisison. Beamforming not executed...\n");
#endif

}

#endif /* CUKERNELS_CUH_ */
