
#include <hip/hip_runtime.h>
#ifdef CUKERNELS_CUH_


/** UTILS **/
__device__ __half2 __hCmul2(__half2 a, __half2 b)
{
		const __half r = a.x * b.x - a.y * b.y;
		const __half i = a.x * b.y + a.y * b.x;

		__half2 val; val.x = r; val.y = i;
		return val;
}


/** STOKES I Beamformer **/
template<typename T=float2, typename U=float>__global__
void simple_bf_tafpt_power(const float2 *idata, float *odata, const float2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
	float2 acc{.0,.0};

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int i = 0; i < conf.n_antenna; i++)
		{
			acc.x = 0; acc.y = 0;
			for(int k = 0; k < conf.n_pol; k++)
			{
				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
		}
	}
}

template<typename T=__half2, typename U=__half>__global__
void simple_bf_tafpt_power(const __half2 *idata, __half *odata, const __half2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	int tidz = blockIdx.z * blockDim.z;								// Channel dimension
	__half2 acc(0,0);

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel + tidx * conf.n_channel;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int i = 0; i < conf.n_antenna; i++)
		{
			acc.x = 0; acc.y = 0;
			for(int k = 0; k < conf.n_pol; k++)
			{
				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + tidz] += (acc.x*acc.x + acc.y*acc.y);
		}
	}
}


template<typename T=float2, typename U=float>__global__
void bf_tafpt_power(const float2 *idata, float *odata, const cudaTextureObject_t weights, const bf_config_t conf)
{
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;      // Time dimension (T)
	int bidy = blockIdx.y;      // Channel dimension (F)

	const int n_elements = conf.n_antenna * conf.n_pol;
	const int warp_idx = tidx / WARP_SIZE;
	const int warp = tidx % WARP_SIZE;


	const int idata_offset = bidx * conf.interval * conf.n_channel * n_elements + bidy * n_elements;
	const int odata_offset = bidy * conf.n_samples / conf.interval + bidx;

	// float2
	float2 voltage;
	float power;
	float xy = 0;

	extern __shared__ char s_mem[];

	float2 *s_idata = reinterpret_cast<float2*>(&s_mem[0]); // Shared memory for input data

	float *s_odata = reinterpret_cast<float*>(&s_mem[n_elements * sizeof(float2)]); // Shared memory for output data

	for(int t = 0; t < conf.interval; t++)
	{

		// load idata to shared memory for one timestep
		for(int i = 0; i < n_elements; i += NTHREAD)
		{
			s_idata[tidx + i] = idata[idata_offset + tidx + i + ];
		}
		//syncthreads();
		// All beams of one timestamps of one channel
		for(int b = 0; b < conf.n_beam; b += WARPS)
		{
			// For each beam set accumulator 'xy' to zero
			xy = 0;
			for(int a = 0; a < conf.n_antenna; a += WARP_SIZE)
			{
				for(int p = 0; p < conf.n_pol; p++)
				{
					// Complex multiplication: raw voltage * weight = voltage
					voltage = cuCmulf(s_idata[conf.n_pol * (a + warp_idx) + p],
						tex3D<float2>(weights, warp * b, bidy, conf.n_pol * (a + warp_idx)));
					// Cacluate (real) power; Square and root cancel each other out
					power = voltage.x * voltage.x + voltage.y * voltage.y;
					// Add weighted power of all elements of one beam to accumulator 'xy'
					xy += power / conf.interval;
				}
			}
			// Every thread accumulated polarizations + n_antenna/WARP_SIZE
			// Load accumulated result to shared memory; Every thread has its own field, otherwise race condition may occur
			s_odata[warp + b + warp_idx] = xy;
		}
	}

	// Reduction
	// __syncthreads();
	for(int b = 0; b < conf.n_beam; b += WARPS)
	{
		int i = WARP_SIZE/2;
		while(i != 0)
		{
			if(warp_idx < i)
				s_odata[warp + b + warp_idx] += s_odata[warp + b + warp_idx + i];
			i /= 2;
		}
		// After reduction the first thread within a warp transfers calculated beams to the global memory
		if(warp_idx == 0)
		{
			odata[odata_offset + b * conf.n_channel * conf.n_samples / conf.interval] = s_odata[warp + b];
		}
	}
}


template<typename T=__half2, typename U=__half>__global__
void bf_tafpt_power(const __half2 *idata, __half *odata, const cudaTextureObject_t weights, const bf_config_t conf)
{
	// int tidx = threadIdx.x + blockIdx.x * blockDim.x;	// Time dimension
	// int tidy = blockIdx.y * blockDim.y;								// Beam dimension
	// int tidz = blockIdx.z * blockDim.z;								// Antenna dimension



}


/** Voltage Beamformer **/


template<typename T=float2>__global__
void simple_bf_tafpt_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
	int tidz = blockIdx.z * blockDim.z;									// Channel dimension

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	float2 acc;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int k = 0; k < conf.n_pol; k++)
		{
			acc = {0,0};
			for(int i = 0; i < conf.n_antenna; i++)
			{
				acc = cuCaddf(acc, cuCmulf(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
					  weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + k] = acc;
		}
	}
}

template<typename T=__half2>__global__
void simple_bf_tafpt_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;		// Time dimension
	int tidy = blockIdx.y * blockDim.y;									// Beam dimension
	int tidz = blockIdx.z * blockDim.z;									// Channel dimension

	int in_offset = tidx * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;
	int out_offset = tidy * conf.n_samples * conf.n_channel * conf.n_pol + tidx * conf.n_channel * conf.n_pol + tidz * conf.n_pol ;
	int weight_offset = tidy * conf.n_antenna * conf.n_channel * conf.n_pol + tidz * conf.n_pol;

	__half2 acc;

	if(tidx < conf.n_samples && tidy < conf.n_beam && tidz < conf.n_channel)
	{
		for(int k = 0; k < conf.n_pol; k++)
		{
			acc = {0,0};
			for(int i = 0; i < conf.n_antenna; i++)
			{
				acc = __hadd2(acc, __hCmul2(idata[in_offset + i * conf.n_channel * conf.n_pol + k],
						weights[weight_offset + i * conf.n_channel * conf.n_pol + k]));
			}
			odata[out_offset + k] = acc;
		}
	}
}


template<typename T=float2>__global__
void bf_tfpat_voltage(const float2 *idata, float2 *odata, const float2 *weights, const bf_config_t conf)
{
	// Grid layout: x = time; y = beam; z = channel
	// Block layout: A block consist of NTHREADS and WARPS. Every warp
	// calculates both polarisations of one beam for one channel at one given time step.
	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
	// Data products are as follow (glob mem):
	// 		idata: TFAP(t)
	//		odata: BFT
	//		weights: BFAP
	// constraints:
	//	- n_antenna must be a multiple of WARP_SIZE 32
	//	- n_samples must be a multiple of WARP_SIZE 32

	int tidx = threadIdx.x;
	int bidx = blockIdx.x;      // Time dimension (T)
	int bidy = blockIdx.y;      // Beam dimension (B)
	int bidz = blockIdx.z;      // Channel dimension (F)

	int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
	int warp = tidx / WARP_SIZE; // Calculate the current warp
	int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index

	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
	int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
	int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
	int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations

	// To enable higher throughput and more efficient data transfer, shared memory
	// is required. The size in bytes of shared memory is calculated as follows:
	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
	extern __shared__ float2 shared_mem_fp32[];	// dynamically allocated
	float2* shared_idata = (&shared_mem_fp32[0]);	// idata space comes first
	float2* shared_weights = (&shared_mem_fp32[n_elements * WARPS]);	// weight space with idata space as offset

	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuComplex).
	float2 __shared__ shared_odata[NTHREAD];

	shared_odata[tidx] = {0,0};	// intialize output with zeros

	float2 acc = {0,0}; // local register for storing intermediate results

	// Load idata and weights into shared memory for every warp
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
		// Here each thread within a warp accesses adjacent samples!
		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];

		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
		// However, this approach prevents race conditions.
		if(warp == 0)
			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
	}

	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.

	// Iterate across all elements.
	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
	// 		FLOP/thread = n * (6+2)
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		acc = cuCaddf(acc, cuCmulf(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx]));
	}
	shared_odata[tidx] = acc;

	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.

	// Since odata contains NTHREAD samples which have not been combined to WARPS time steps a reduction is required.
	int i = WARP_SIZE / 2;
	// This reduction may no be very efficient since many threads within a warp are idled
#pragma unroll
	while(i != conf.n_pol - 1)
	{
		if(warp_idx < i)
			shared_odata[tidx] = cuCaddf(shared_odata[tidx], shared_odata[tidx + i]);
		__syncthreads();
		i /= 2;
	}
	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
	// samples are transfered back to the global memory for each block.
	if(warp_idx < conf.n_pol)
	{
		// TODO: In case of integer inputs, conversion is implemented here!!!
		odata[output_glob_idx] = shared_odata[tidx];	// Polarisation 0 and 1
	}

}


template<typename T=__half2>__global__
void bf_tfpat_voltage(const __half2 *idata, __half2 *odata, const __half2 *weights, const bf_config_t conf)
{

	// Grid layout: x = time; y = beam; z = channel
	// Block layout: A block consist of 1024 threads and 32 warps. Every warp
	// calculates both polarisations of one beam for one channel at one given time step.
	// Within a block 32 adjacent time steps for one beam are calculated (same channel).
	// Data products are as follow (glob mem):
	// 		idata: TFAP(t)
	//		odata: BFT
	//		weights: BFAP
	// constraints:
	//	- n_antenna must be a multiple of WARP_SIZE 32
	//	- n_samples must be a multiple of WARP_SIZE 32

	const int tidx = threadIdx.x;
	const int bidx = blockIdx.x;      // Time dimension (T)
	const int bidy = blockIdx.y;      // Beam dimension (B)
	const int bidz = blockIdx.z;      // Channel dimension (F)
#if __CUDA_ARCH__ >= 530
	const int n_elements = conf.n_antenna * conf.n_pol; // Number of elements, product of antenna (A) and polarisation (P)
	const int warp = tidx / WARP_SIZE; // Calculate the current warp
	const int warp_idx = tidx % WARP_SIZE;    // thread index -> warp index

	// Each thread has its own indices for accessing the global memory (idata, odata, weights).
	const int idata_glob_idx = warp_idx + n_elements * (bidx * WARPS * conf.n_channel + warp * conf.n_channel + bidz);
	const int weights_glob_idx = warp_idx + n_elements * (bidy * conf.n_channel + bidz);
	const int output_glob_idx = (bidy * conf.n_samples * conf.n_channel
			+ bidz * conf.n_samples + bidx * WARPS + warp)*conf.n_pol + warp_idx; // multiplied by two, since two polarisations

	// To enable higher throughput and more efficient data transfer, shared memory
	// is required. The size in bytes of shared memory is calculated as follows:
	//	shared_mem_bytes = sizeof(T) * (A * P * (WARPS + 1) + NTHREADS)
	//	idata  = sizeof(T) * A * P * WARPS	<- every warp loads data of all elements at one time step
	// 	weight = sizeof(T) * A * P							<- weights are the same for all warps
	//  odata  = sizeof(T) * NTHREADS						<- Every thread calculates one output sample
	extern __shared__ __half2 shared_mem_fp16[];	// dynamically allocated
	__half2* shared_idata = (&shared_mem_fp16[0]);	// idata space comes first
	__half2* shared_weights = (&shared_mem_fp16[n_elements * WARPS]);	// weight space with idata space as offset

	// To prevent overflows when using integer values, the datatype of shared_odata has to be float2 (cuCOmplex).
	__half2 __shared__ shared_odata[NTHREAD];

	shared_odata[tidx] = {0,0};	// intialize output with zeros

	__half2 acc = {0,0}; // local register for storing intermediate results

	// Load idata and weights into shared memory for every warp
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		// It is important to access 32 adjacent samples, to increase the hit rate of cached memory
		// Here each thread within a warp accesses adjacent samples!
		shared_idata[warp * n_elements + i + warp_idx] = idata[idata_glob_idx + i];

		// Since all warps within a block are using the same weights, only one warp needs to load the weights.
		// This may not be the most efficient way, since all other 31 warps are idled until the weights are loaded.
		// However, this approach prevents race conditions.
		if(warp == 0)
			shared_weights[i + warp_idx] = weights[weights_glob_idx + i];
	}

	__syncthreads();	// Synchronize all threads within a block, to ensure all data is loaded.

	// Iterate across all elements.
	// Each thread within a warp performs n complex multiplications and 2*n additions (n = n_elements/WARP_SIZE).
	// 		FLOP/thread = n * (6+2)
#pragma unroll
	for(int i = 0; i < n_elements; i+=WARP_SIZE)
	{
		acc = __hadd2(acc, (__hCmul2(shared_idata[warp * n_elements + i + warp_idx], shared_weights[i + warp_idx])));
	}
	shared_odata[tidx] = (acc);

	__syncthreads(); // Synchronize all threads within a block, to ensure all computitations are done.

	// Since odata contains 1024 samples which have not been combined to 32 time steps a reduction is required.
	int i = WARP_SIZE / 2;
	// This reduction may no be very efficient since many threads within a warp are idled
#pragma unroll
	while(i != conf.n_pol - 1)
	{
		if(warp_idx < i)
			shared_odata[tidx] = __hadd2(shared_odata[tidx], shared_odata[tidx + i]);
		__syncthreads();
		i /= 2;
	}

	// After reduction the first two samples in shared_odata with warp offset contains both polarisations.
	// So, if warp_idx is 0 or 1, assign the samples to the global output buffer. In total 64
	// samples are transfered back to the global memory for each block.
	if(warp_idx < conf.n_pol)
	{
		// TODO: In case of integer inputs, conversion is implemented here!!!
		odata[output_glob_idx] = (shared_odata[tidx]);	// Polarisation 0 and 1
	}

#else
if(tidx == 0 && bidx==0 && bidy == 0 && bidz == 0)
	printf("Warning: CUDA architecture does not support half precisison. Beamforming not executed...\n");
#endif

}

#endif /* CUKERNELS_CUH_ */
